#include "hip/hip_runtime.h"
#include "vecAdd.h"

#include <algorithm>
#include <stdexcept>

using namespace std;

namespace {

__global__ void _vecAdd(
    const double * a,
    const double * b,
    double * c,
    size_t maxIdx
)
{
    size_t idx = gridDim.x * blockIdx.x + threadIdx.x;
    if(idx < maxIdx) {
        c[idx] = a[idx] + b[idx];
    }
}

} // namespace


void vecAdd(
    const std::vector<double>& a,
    const std::vector<double>& b,
    std::vector<double>& result
)
{
    if(a.size() != b.size()) {
        throw std::logic_error("Vectors must have same size");
    }
    result.resize(a.size());

    // Query current device
    int device;
    hipGetDevice(&device);

    // Query device properties
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, device);

    // Assuming that all vectors can fit into GPU's memory!
    const size_t vectorSize = a.size() * sizeof(double);

    double* deviceA;
    hipMalloc(&deviceA, vectorSize);
    hipMemcpy(
	deviceA,
	a.data(),
	vectorSize,
	hipMemcpyHostToDevice
    );

    double* deviceB;
    hipMalloc(&deviceB, vectorSize);
    hipMemcpy(
	deviceB,
	b.data(),
	vectorSize,
	hipMemcpyHostToDevice
    );

    double* deviceC;
    hipMalloc(&deviceC, vectorSize);

    // Assuming that vectors can be added via a single grid!
    size_t numThreads = min((size_t)props.maxThreadsDim[0], result.size());
    size_t numBlocks = (result.size() - 1) / numThreads + 1;

    _vecAdd<<<numBlocks, numThreads>>>(deviceA, deviceB, deviceC, result.size());
    hipDeviceSynchronize();

    hipMemcpy(
        result.data(),
	deviceC,
	vectorSize,
	hipMemcpyDeviceToHost
    );

    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);
}

